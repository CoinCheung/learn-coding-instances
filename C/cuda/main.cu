
#include<iostream>
#include <vector>
#include <numeric>
#include <cmath>
#include<hip/hip_runtime.h>


using std::cout;
using std::endl;
using std::vector;



template<typename scalar_t>
__global__ void compute_square(const int num, const scalar_t* data, scalar_t* res, float* time) {
    clock_t start = clock();
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i{tid}; i < num; i+=stride) {
        res[i] = data[i] * data[i];
    }

    clock_t finish = clock();
    if (tid < num) {
        time[tid] = (float)(finish - start) / CLOCKS_PER_SEC;
    }
}


template<typename scalar_t>
__global__ void compute_sum(const int num, const scalar_t* data, scalar_t* sum) {
    // dynamic allocated
    extern __shared__ __align__(sizeof(scalar_t)) int shared_raw[]; 
    scalar_t *shared = reinterpret_cast<scalar_t*>(shared_raw);
    shared[threadIdx.x] = 0; // assign 0 to the aligned memory
    __syncthreads();

    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < num) {
        shared[threadIdx.x] = data[tid];
    }
    __syncthreads();

    /* int mask = 1, offset = 1;
     * while (offset < blockDim.x) {
     *     if ((tid & mask) == 0) {
     *         shared[threadIdx.x] += shared[threadIdx.x + offset];
     *     }
     *     offset += offset;
     *     mask += offset;
     *     __syncthreads();
     * } */

    // same as above
    for (int s=1; s < blockDim.x; s*=2) {
        int idx = 2 * s * threadIdx.x;
        if (idx < blockDim.x) {
            shared[idx] += shared[idx + s];
        }
        __syncthreads();
    }

    // faster: but only support blockdim is 2**x
    /* for (unsigned int s{blockDim.x / 2}; s > 0; s>>=1) { */
    /*     int idx = threadIdx.x; */
    /*     if (idx < s) { */
    /*         shared[idx] += shared[idx + s]; */
    /*     } */
    /*     __syncthreads(); */
    /* } */

    if (threadIdx.x == 1) { // must be one thread, or will be added many times
        atomicAdd(sum, shared[0]);
    }
}


void check_gpu_info(int device_id) {
    int dev = device_id;
    hipDeviceProp_t devProp;
    if (hipGetDeviceProperties(&devProp, dev)) {
        std::cout << "get properties fail" << std::endl;
        return;
    }
    cout << "device " << dev << ": " << devProp.name << endl;
    cout << "num of SM: " << devProp.multiProcessorCount << endl;
    cout << "shared memory of each block: " 
        << devProp.sharedMemPerBlock
        << endl;
    cout << "max thread num of each block: "
        << devProp.maxThreadsPerBlock
        << endl;
    cout << "max thread num of each SM: "
        << devProp.maxThreadsPerMultiProcessor
        << endl;
    cout << "max thread wrap num of each SM: "
        << devProp.maxThreadsPerMultiProcessor / 32
        << endl;

}

bool InitCUDA() {
    int count;
    hipGetDeviceCount(&count);
    if (count == 0) {cout << "no gpu device\n"; return false;}
    for (int i{0}; i < count; ++i) {
        check_gpu_info(i);
    }
    hipSetDevice(0);
    return true;
}


void test_cuda_sync() {
    cout << "test sync" << endl;
    int len{1000};
    vector<float> data(len);
    vector<float> res(len); 
    vector<float> time(len);
    float sum{0};
    std::iota(data.begin(), data.end(), 0);

    // allocate memory
    float *dev_data{nullptr}, *dev_res{nullptr}, *dev_time{nullptr};
    float *dev_sum{nullptr};
    hipMalloc((void**)&dev_data, sizeof(float) * len);
    hipMalloc((void**)&dev_res, sizeof(float) * len);
    hipMalloc((void**)&dev_time, sizeof(float) * len);
    hipMalloc((void**)&dev_sum, sizeof(float));

    // copy data to device
    hipMemcpy(dev_data, data.data(), sizeof(float) * len, hipMemcpyHostToDevice);
    hipMemcpy(dev_sum, &sum, sizeof(float), hipMemcpyHostToDevice);

    // execute kernel function
    dim3 block(512);
    dim3 grid(std::min(4096, (int)std::ceil(len / 512.))); 
    compute_square<float><<<grid, block, 4096>>>(len, dev_data, dev_res, dev_time);
    compute_sum<float><<<grid, block, 4096>>>(len, dev_data, dev_sum);

    /// test double type when instantiate template with two types
    // 1. must add compilation option of -arch=sm_60 to support double
    // 2. if to use dynamic shared memory, take care of its declaration
    vector<double> datad(len);
    vector<double> resd(len);
    double *dev_datad{nullptr};
    double *dev_sumd{nullptr};
    hipMalloc((void**)&dev_datad, sizeof(double) * len);
    hipMalloc((void**)&dev_sumd, sizeof(double));
    compute_sum<double><<<grid, block, 4096>>>(len, dev_datad, dev_sumd);

    // copy results back to host
    hipMemcpy(&res[0], dev_res, sizeof(int) * len, hipMemcpyDeviceToHost);
    hipMemcpy(&time[0], dev_time, sizeof(float) * len, hipMemcpyDeviceToHost);
    hipMemcpy(&sum, dev_sum, sizeof(int), hipMemcpyDeviceToHost);
    vector<int> tmp_cpu(len);

    // free allocated memory
    hipFree(dev_data);
    hipFree(dev_res);
    hipFree(dev_time);
    hipFree(dev_sum);

    // check results
    cout << "cuda reduce sum is: \n";
    cout << sum << endl;
    sum = 0;
    for (auto &el : data)
        sum += el;
    cout << "cpu sum is: \n";
    cout << sum << endl;

    for (int i{0}; i < 10; ++i) {
        cout << res[i] << ", ";
    } cout << endl;
    cout << (float)time[0] << endl;

}


void test_cuda_async() {
    cout << "test async method\n";
    const int len = 1000;

    // 异步操作是通过流来维护的
    // 一个流对应一个顺序提交执行的任务，多个流多个任务，
    // 而且不同流之间还可以并行  
    // 建立并初始化流 
    hipStream_t stream[2];
    for (int i{0}; i < 2; ++i) {
        hipStreamCreate(&stream[i]);
    }

    // 异步操作只能通过锁页内存来交换数据，不能直接用malloc或者栈上的内存
    // cpu上分配锁页内存, 有三种方法: 
    float *data, *res;
    vector<float> time(len);
    float sum{0};
    hipError_t status;
    // 1. 最简单的分配方法
    status = hipHostMalloc(&data, sizeof(float) * len, hipHostMallocDefault);
    if (status != hipSuccess) {
        cout << "fail to allocate pinned memory\n";
    }
    // 2. 带选项的分配方法
    status = hipHostAlloc(&res, sizeof(float) * len, hipHostMallocDefault);
    if (status != hipSuccess) {
        cout << "fail to allocate pinned memory\n";
    }
    // 3. 把已有的可分页内存注册成锁负内存
    status = hipHostRegister(&time[0], sizeof(float) * len, hipHostRegisterDefault);
    if (status != hipSuccess) {
        cout << "fail to register as pinned memory\n";
    }
    status = hipHostRegister(&sum, sizeof(float), hipHostRegisterDefault);
    std::iota(data, data+len, 0);


    // 像同步一样，也要分配gpu全局内存  
    float *dev_data1{nullptr}, *dev_res{nullptr}, *dev_time{nullptr};
    float *dev_data2{nullptr}, *dev_sum{nullptr};
    hipMalloc((void**)&dev_data1, sizeof(float) * len);
    hipMalloc((void**)&dev_res, sizeof(float) * len);
    hipMalloc((void**)&dev_time, sizeof(float) * len);
    hipMalloc((void**)&dev_data2, sizeof(float) * len);
    hipMalloc((void**)&dev_sum, sizeof(float));

    // 定义并行参数
    dim3 block(512);
    dim3 grid(std::min(4096, (int)std::ceil(len / 512.)));

    // 使用流和锁页内存异步调用核函数
    // 为了提高效率最好两个流交替执行，先一起拷数据，再一起运行核函数
    // 1. 使用异步方式把数据拷贝到gpu上去
    status = hipMemcpyAsync(dev_data1, data, sizeof(float) * len, hipMemcpyHostToDevice, stream[0]);
    status = hipMemcpyAsync(dev_data2, data, sizeof(float) * len, hipMemcpyHostToDevice, stream[1]);
    // 2. 调用核函数，加上stream就是异步的
    compute_square<float><<<grid, block, 0, stream[0]>>>(len, dev_data1, dev_res, dev_time);
    compute_sum<float><<<grid, block, 2048, stream[1]>>>(len, dev_data2, dev_sum);
    // 3. 执行完再拷回来，只是一个请求，异步的，nonblock的
    status = hipMemcpyAsync(&res[0], dev_res, sizeof(float) * len, hipMemcpyDeviceToHost, stream[0]);
    status = hipMemcpyAsync(&sum, dev_sum, sizeof(float), hipMemcpyDeviceToHost, stream[1]);


    // 等待stream处理完所有的任务 
    // 上面只是向stream提交了请求，运行到这一步的时候不一定已经完成了，所以要在这等上面的stream任务都完成
    for (int i{0}; i < 2; ++i) {
        hipStreamSynchronize(stream[i]);
    }

    // 检查结果  
    cout << "square: \n";
    for (int i{0}; i < 10; ++i) {
        cout << res[i] << ", ";
    }cout << endl;
    cout << "sum: \n";
    cout << sum << endl;

    // 最后还得回收上面分配的各种资源  
    // 回收上面分配的锁页内存  
    hipHostFree(data);
    hipHostFree(res);
    hipHostUnregister(&time[0]);
    hipHostUnregister(&sum);
    // 回收gpu的内存
    hipFree(dev_data1);
    hipFree(dev_res);
    hipFree(dev_time);
    hipFree(dev_data2);
    // 销毁上面分配的流
    for (int i{0}; i < 2; ++i) {
        hipStreamDestroy(stream[i]);
    }
}


int main(void) {
    if (!InitCUDA()) {
        return 0;
    }
    test_cuda_sync();
    test_cuda_async();
    return 0;
}
