
#include<iostream>
#include <vector>
#include <numeric>
#include <cmath>
#include<hip/hip_runtime.h>


using std::cout;
using std::endl;
using std::vector;



template<typename scalar_t>
__global__ void compute_squ_sum(const int num, const scalar_t* data, scalar_t* res, scalar_t* sum, clock_t* time) {
/* __global__ void compute_squ_sum(const int num, scalar_t* res) { */
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int th_idx = bid * blockDim.x + tid;
    int stride = blockDim.x * gridDim.x;
    for (int i{th_idx}; i < num; i+=stride) {
        res[i] = data[i] * data[i];
        res[i] = 10;
    }
    res[0]=111;
    res[1]=111;
}

void check_gpu_info(int device_id) {
    int dev = device_id;
    hipDeviceProp_t devProp;
    if (hipGetDeviceProperties(&devProp, dev)) {
        std::cout << "get properties fail" << std::endl;
        return;
    }
    cout << "device " << dev << ": " << devProp.name << endl;
    cout << "num of SM: " << devProp.multiProcessorCount << endl;
    cout << "shared memory of each block: " 
        << devProp.sharedMemPerBlock
        << endl;
    cout << "max thread num of each block: "
        << devProp.maxThreadsPerBlock
        << endl;
    cout << "max thread num of each SM: "
        << devProp.maxThreadsPerMultiProcessor
        << endl;
    cout << "max thread wrap num of each SM: "
        << devProp.maxThreadsPerMultiProcessor / 32
        << endl;

}

bool InitCUDA() {
    int count;
    hipGetDeviceCount(&count);
    if (count == 0) {cout << "no gpu device\n"; return false;}
    for (int i{0}; i < count; ++i) {
        check_gpu_info(i);
    }
    hipSetDevice(0);
    return true;
}


void test_cuda_sync() {
    cout << "test sync" << endl;
    int len{1000};
    vector<int> inp(len);
    /* vector<int> res(len); */ // this should not do
    int res[len];
    std::iota(inp.begin(), inp.end(), 0);

    for (int i{0};i<1000; ++i) res[i] = i;

    // allocate memory
    int *data{nullptr}, *squ{nullptr};
    int sum{0};
    clock_t *time;
    hipMalloc((void**)&data, sizeof(int) * len);
    hipMalloc((void**)&squ, sizeof(int) * len);
    hipMalloc((void**)&time, sizeof(clock_t) * len);

    // copy data to device
    hipMemcpy(data, inp.data(), sizeof(int) * len, hipMemcpyHostToDevice);

    // execute kernel function
    dim3 block(512);
    dim3 grid(std::ceil(len / 512.));
    compute_squ_sum<int><<<32, 512, 0>>>(len, data, squ, &sum, time);
    /* compute_squ_sum<int><<<32, 512, 0>>>(len, squ); */

    // copy results back to host
    hipMemcpy(&res, squ, sizeof(int) * len, hipMemcpyDeviceToHost);

    // free allocated memory
    hipFree(data);
    hipFree(squ);
    hipFree(time);

    cout << sum << endl;
    for (int i{0}; i < 10; ++i) {
        cout << res[i] << ", ";
    } cout << endl;
}

int main(void) {
    if (!InitCUDA()) {
        return 0;
    }
    // kernel<<<2, 2>>>();
    // check_gpu_info();
    test_cuda_sync();
    return 0;
}
